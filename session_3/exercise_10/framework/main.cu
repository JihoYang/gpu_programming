#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

// Exercise 10

// Written by: Jiho Yang (M.Sc student in Computational Science & Engineering)
// Matriculation number: 03675799

#include "helper.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <unistd.h>
using namespace std;

const float pi = 3.141592653589793238462f;

// uncomment to use the camera
//#define CAMERA

// Compute gradient
__global__ void compute_gradient(float *d_gradx, float *d_grady, float *d_imgIn, int w, int h, int nc){
	// Get x y z pixel coordinates in 3D kernel
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int z = threadIdx.z + blockIdx.z*blockDim.z;
	// Get high indices
	size_t x_high = x + 1 + (size_t)w*y + (size_t)h*w*z;
	size_t y_high = x + (size_t)w*(y+1) + (size_t)h*w*z;
	size_t idx = x + (size_t)w*y + (size_t)h*w*z;
	// Ensure no threads are out of problem domain
	if (x < w && y < h && z < nc){
	// Compute gradient
		if (x < w-1){
			d_gradx[idx] = d_imgIn[x_high] - d_imgIn[idx];
		} else
			d_gradx[idx] = 0;
		if (y < h-1){
			d_grady[idx] = d_imgIn[y_high] - d_imgIn[idx];
		} else
			d_grady[idx] = 0;
	}
}

// Compute L2 norm
__device__ void compute_norm(float *d_norm, float *d_vec1, float *d_vec2, int w, int h, int nc){
	// Temporary variable for norm
	float sqrd1 = 0;
	float sqrd2 = 0;
	float val1, val2;
	// Get coordinates
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	// Get index
	int idx = x + (size_t)w*y;
	// Compute norm
	if (x < w && y < h){
		for (size_t c = 0; c < nc; c++){
			// Get index
			size_t idx_3d = idx + (size_t)w*h*c;
			// Compute L2 norm
			val1 = d_vec1[idx_3d];
			val2 = d_vec2[idx_3d];
			sqrd1 += val1*val1;
			sqrd2 += val2*val2;
		}
		d_norm[idx] = sqrtf(sqrd1*sqrd1 + sqrd2*sqrd2);
	}
}

// Compute divergence
__global__ void compute_divergence(float *d_div, float *d_gradx, float *d_grady, int w, int h, int nc){
	// Get x y z pixel coordinates in 3D kernel
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int z = threadIdx.z + blockIdx.z*blockDim.z;
	// Get low indices
	size_t idx = x + (size_t)w*y + (size_t)h*w*z;
	size_t x_low = x-1 + (size_t)w*y + (size_t)h*w*z;
	size_t y_low = x + (size_t)w*(y-1) + (size_t)h*w*z;
	// Temporary values 
	float v_x, v_y;
	// Ensure no threads are out of problem domain
	if (x < w && y < h && z < nc){
		// Compute divergence
		if (x > 1){
			v_x = d_gradx[idx] - d_gradx[x_low];
			
		} else
			v_x = 0;
		if (y > 1){
			v_y = d_grady[idx] - d_grady[y_low];
		} else
			v_y = 0;
		// Sum gradients
		d_div[idx] = v_x + v_y;
	}

	if (idx == 100){
		printf("Divergence = %f\n", d_div[idx]);
	}
}
	
// Convolution on global memory
__global__ void convolution_global(float *d_imgIn, float *d_imgOut, float *d_kernel, int w, int h, int nc, int w_kernel, int h_kernel){
	// Get coordinates
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	//int z = threadIdx.z + blockDim.z*blockIdx.z;
	// Get indices
	size_t idx = x + (size_t)w*y;
	// Initialise d_imgOut
	// Set origin
	int mid = (w_kernel-1)/2;
	// Convolution - Note x_kernel is the global x coordinate of kernel in the problem domain
	for (size_t c = 0; c < nc; c++){
		size_t idx_3d = idx + (size_t)w*h*c;
		d_imgOut[idx_3d] = 0.0f;
		if (x < w && y < h){
			for (size_t j = 0; j < h_kernel; j++){
				for (size_t i = 0; i < w_kernel; i++){
					// Boundary condition
					int x_kernel_global = x - mid + i;
					int y_kernel_global = y - mid + j;
					// clamping
					if (x_kernel_global < 0){
						x_kernel_global = 0;
					}
					if (x_kernel_global > w-1){
						x_kernel_global = w - 1;
					}
					if (y_kernel_global < 0){
						y_kernel_global = 0;
					}
					if (y_kernel_global > h - 1){
						y_kernel_global = h - 1;
					}
					// Get indices
					int idx_kernel_local = i + w_kernel*j;
					int idx_kernel_global = x_kernel_global + w*y_kernel_global + w*h*c;
					// Multiply and sum
					d_imgOut[idx_3d] += d_kernel[idx_kernel_local] * d_imgIn[idx_kernel_global];
				}
			}
		}
	}
}

// Set up kernel
void get_kernel(float *kernel, int w_kernel, int h_kernel, const float pi, float sigma){
	//Set up parameters
	int origin = w_kernel/2;
	float total = 0.0f;
	// Define 2D Gaussian kernel
	for (size_t y_kernel = 0; y_kernel < h_kernel; y_kernel++){
		for (size_t x_kernel = 0; x_kernel < w_kernel; x_kernel++){
			int a = x_kernel - origin;
			int b = y_kernel - origin;
			int idx = x_kernel + w_kernel*y_kernel;
			kernel[idx] = (1.0f / (2.0f*pi*sigma*sigma))*exp(-1*((a*a+b*b) / (2*sigma*sigma)));
			total += kernel[idx];
		}
	}
	// Normalise kernel
	for (size_t y_kernel = 0; y_kernel < h_kernel; y_kernel++){
		for (size_t x_kernel = 0; x_kernel < w_kernel; x_kernel++){
			int idx = x_kernel + w_kernel*y_kernel;
			kernel[idx] /= total;
		}
	}
}

// Compute eigenvalue of a 2 by 2 matrix
__device__ void compute_eigenvalue(float &eigen_value_0, float &eigen_value_1, float &eigen_vector_0, float &eigen_vector_1, float &eigen_vector_2, float &eigen_vector_3, float d_t1_val, float d_t2_val, float d_t3_val, int w, int h){
	// Define matrix	
	float A[4] = {d_t1_val, d_t2_val, d_t2_val, d_t3_val};
	// Define elements
	float a = A[0];
	float b = A[1];
	float c = A[2];
	float d = A[3];	
	// Trace and determinant
	float T = a + d;
	float D = a*d - b*c;
	// Compute eigenvalue
	eigen_value_0 = T/2 + sqrtf(T*T/4-D);
	eigen_value_1 = T/2 - sqrtf(T*T/4-D);
	// Sort eigenvalue array (val_1 > val_2)
	if (eigen_value_0 < eigen_value_1){
		float swap = eigen_value_0;
		eigen_value_0 = eigen_value_1;
		eigen_value_1 = swap;
	}
	// Compute eigenvectors
	/*
	if (c != 0){
		eigen_vector_0 = eigen_value_0 - d;
		eigen_vector_1 = c;
		eigen_vector_2 = eigen_value_1 - d;
		eigen_vector_3 = c;
	}
	*/
	else if (b != 0){
		eigen_vector_0 = b;
		eigen_vector_1 = eigen_value_0 - a;
		eigen_vector_2 = b;
		eigen_vector_3 = eigen_value_1 - a;
	}
	else if (b == 0 && c == 0){
		eigen_vector_0 = 1;
		eigen_vector_1 = 0;
		eigen_vector_2 = 0;
		eigen_vector_3 = 1;
	}

	// Scale eigenvector
	eigen_vector_0 = 1*eigen_vector_0;	
	eigen_vector_1 = 1*eigen_vector_1;	
	eigen_vector_2 = 1*eigen_vector_2;	
	eigen_vector_3 = 1*eigen_vector_3;	

	// Get coordinates
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int z = threadIdx.z + blockDim.z*blockIdx.z;
	// Get index
	size_t idx = x + (size_t)w*y + (size_t)w*h*z;
/*
	if (idx == 0){
		printf("a = %f\n", a);
		printf("b = %f\n", b);
		printf("c = %f\n", c);
		printf("d = %f\n", d);
	}
*/
}

// Apply anisotropic diffusion
__global__ void apply_diffusion(float *d_gradx, float *d_grady, float *d_imgIn, float alpha, float C, float *d_t1, float *d_t2, float *d_t3, int w, int h, int nc){
	// Define eigenvalue and eigenvector
	float eigen_value_0, eigen_value_1;
	float eigen_vector_0, eigen_vector_1, eigen_vector_2, eigen_vector_3;
	// Get coordinates
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int z = threadIdx.z + blockDim.z*blockIdx.z;
	// Get index
	size_t idx = x + (size_t)w*y + (size_t)w*h*z;
	size_t idx_2d = x + (size_t)w*y;
	// Compute eigenvalues and eigenvector
	if (x < w && y < h && z < nc){
		compute_eigenvalue(eigen_value_0, eigen_value_1, eigen_vector_0, eigen_vector_1, eigen_vector_2, eigen_vector_3, d_t1[idx_2d], d_t2[idx_2d], d_t3[idx_2d], w, h);
	}
	__syncthreads();
	// Get Mu
	float mu_1 = alpha;
	float mu_2;
	if (eigen_value_0 == eigen_value_1){
		mu_2 = alpha;
	} else{
		float eigdif = eigen_value_0 - eigen_value_1;
		float inside = -C/(eigdif*eigdif);
		mu_2 =  alpha + (1 - alpha)*exp(inside);
	}
	// Get diffusion tensor
	float G[4];
	G[0] = mu_1*eigen_vector_0*eigen_vector_0 + mu_2*eigen_vector_2*eigen_vector_2;
	G[1] = mu_1*eigen_vector_0*eigen_vector_1 + mu_2*eigen_vector_2*eigen_vector_3;
	G[2] = mu_1*eigen_vector_1*eigen_vector_0 + mu_2*eigen_vector_3*eigen_vector_2;
	G[3] = mu_1*eigen_vector_1*eigen_vector_1 + mu_2*eigen_vector_3*eigen_vector_3;

	G[0] = 1;
	G[1] = 0;
	G[2] = 1;
	G[3] = 0;

	__syncthreads();

/*
	if (idx == 0){
		printf("Before diffusion\n");
		printf("d_gradx = %f\n", d_gradx[idx]);
		printf("d_grady = %f\n", d_grady[idx]);
	}
*/

	// Update gradient
	if (x < w && y < h && z < nc){
		d_gradx[idx] = G[0]*d_gradx[idx] + G[1]*d_grady[idx];
		d_grady[idx] = G[2]*d_gradx[idx] + G[3]*d_grady[idx];
	}
	
	if (idx == 100){
		printf("After diffusion\n");
		printf("d_gradx = %f\n", d_gradx[idx]);
		printf("d_grady = %f\n", d_grady[idx]);


/*
		printf("G[0] = %f\n", G[0]);
		printf("G[1] = %f\n", G[1]);
		printf("G[2] = %f\n", G[2]);
		printf("G[3] = %f\n", G[3]);

		printf("eigenvalue_0 = %f\n", eigen_value_0);
		printf("eigenvalue_1 = %f\n", eigen_value_1);
		printf("eigenvector_0 = %f\n", eigen_vector_0);
		printf("eigenvector_1 = %f\n", eigen_vector_1);
		printf("eigenvector_2 = %f\n", eigen_vector_2);
		printf("eigenvector_3 = %f\n", eigen_vector_3);

		printf("Mu_1 = %f\n", mu_1);
		printf("Mu_2 = %f\n", mu_2);
*/
	}
}

// Update image
__global__ void update_image(float *d_imgIn, float *d_div, float tau, int w, int h, int nc){
	// Get coordinates	
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int z = threadIdx.z + blockDim.z*blockIdx.z;
	// Get index
	size_t idx = x + (size_t)w*y + (size_t)w*h*z;

	if (idx == 100){
		printf("Before update\n");
		printf("d_imgIn = %f\n", d_imgIn[idx]);
	}


	if (x < w && y < h && z < nc){	
		// Update image	
		d_imgIn[idx] += tau * d_div[idx];
	}

	if (idx == 100){
		printf("After update\n");
		printf("d_imgIn = %f\n", d_imgIn[idx]);
	}

}

// Compute M
__global__ void compute_M(float *d_m1, float *d_m2, float *d_m3, float *d_gradx, float *d_grady, int w, int h, int nc){
	// Get coordinates
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	// Get index in matrices m
	size_t idx_2d = x + (size_t)w*y;
	// Initialise sums
	float sum1 = 0;	
	float sum2 = 0;
	float sum3 = 0;
	if (x < w && y < h){
		// Loop through channels
		for (size_t c = 0 ; c < nc; c++){
			// Get index
			size_t idx = x + (size_t)w*y + (size_t)w*h*c;	
			sum1 += d_gradx[idx] * d_gradx[idx];
			sum2 += d_gradx[idx] * d_grady[idx];
			sum3 += d_grady[idx] * d_grady[idx];
		}
		// Fill matrices
		d_m1[idx_2d] = sum1;
		d_m2[idx_2d] = sum2;
		d_m3[idx_2d] = sum3;
	}
}

// Rotationally robust gradient
__global__ void rotational_gradient(float *d_imgIn, float *d_gradx, float *d_grady, int w, int h, int nc){
	// Get coordinates
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int z = threadIdx.z + blockDim.z*blockIdx.z;
	// Get indices
	size_t idx = x + (size_t)w*y + (size_t)w*h*z;
	// Compute gradient
	if (x < w && y < h && z < nc){
		// Get neighbouring indices
		int x_high = x + 1;
		int y_high = y + 1;
		int x_low  = x - 1;
		int y_low  = y - 1;
		// Clamping
		if (x_high > w - 1){
			x_high = w - 1;
		}
		if (y_high > h - 1){
			y_high = h - 1;
		}
		if (x_low < 0){
			x_low = 0;	
		}
		if (y_low < 0){
			y_low = 0;
		}
		// Get indices of neighbouring indices
		size_t idx_x_high_y_high = x_high + (size_t)w*y_high + (size_t)w*h*z;
		size_t idx_x_high_y_low  = x_high + (size_t)w*y_low  + (size_t)w*h*z;
		size_t idx_x_low_y_high  = x_low  + (size_t)w*y_high + (size_t)w*h*z;
		size_t idx_x_low_y_low   = x_low  + (size_t)w*y_low  + (size_t)w*h*z;
		size_t idx_x_high_y_mid  = x_high + (size_t)w*y      + (size_t)w*h*z;
		size_t idx_x_low_y_mid   = x_low  + (size_t)w*y      + (size_t)w*h*z;
		size_t idx_x_mid_y_high  = x      + (size_t)w*y_high + (size_t)w*h*z;
		size_t idx_x_mid_y_low   = x      + (size_t)w*y_low  + (size_t)w*h*z;
		// Compute gradient	
		d_gradx[idx] = (3*d_imgIn[idx_x_high_y_high] + 10*d_imgIn[idx_x_high_y_mid] 
			  		  + 3*d_imgIn[idx_x_high_y_low]  - 3*d_imgIn[idx_x_low_y_high]  
					  - 10*d_imgIn[idx_x_low_y_mid]  - 3*d_imgIn[idx_x_low_y_low])/32;

		d_grady[idx] = (3*d_imgIn[idx_x_high_y_high] + 10*d_imgIn[idx_x_mid_y_high] 
					  + 3*d_imgIn[idx_x_low_y_high]  - 3*d_imgIn[idx_x_high_y_low]  
					  - 10*d_imgIn[idx_x_mid_y_low]  - 3*d_imgIn[idx_x_low_y_low])/32;
	}
}


// main
int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;



	// Diffusion
	float tau = 0.02f;
	int	N = 500;
	// Convolution kernel
	float sigma = 0.5f;
	float phi = 3.0f;
	getParam("sigma", sigma, argc, argv);
	cout << "sigma: " << sigma << endl;
    // ### Define your own parameters here as needed    


    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
	// Define kernel dimensions
	int r = ceil(3*sigma);
	int w_kernel = r * 2 + 1;	  //windowing
	int h_kernel = w_kernel;  	  //Square kernel
	int r_phi = ceil(3*phi);
	int w_kernel_phi = r_phi * 2 + 1;
	int h_kernel_phi = w_kernel_phi;
	// Kernel information
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer


    // ### Define your own output images here as needed
	cv:: Mat mgradx(h, w, mIn.type());
	cv:: Mat mgrady(h, w, mIn.type());
	cv:: Mat mdiv(h, w, mIn.type());

	cv:: Mat mt1(h, w, CV_32FC1);
	cv:: Mat mt2(h, w, CV_32FC1);
	cv:: Mat mt3(h, w, CV_32FC1);



    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

	// Get array memory
	int nbytes = w * h * nc * sizeof(float);
	int nbytes_kernel = w_kernel * h_kernel * sizeof(float);
	int nbytes_kernel_phi = w_kernel_phi * h_kernel_phi * sizeof(float);
    // allocate raw input image array
    float *imgIn = new float[(size_t)nbytes];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);


    // ###
    // ###
    // ### TODO: Main computation
    // ###
    // ###

		
	////////////////////////////////////////////////////////////////////// Block setting ///////////////////////////////////////////////////////////////////////

	dim3 block = dim3(128, 1, 1); 
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (nc + block.z - 1));


	float alpha = 0.01f;
	float C = 0.0000005f;

	Timer timer; timer.start();

	////////////////////////////////////////////////////////////////////// Arrays (Device) /////////////////////////////////////////////////////////////////////

	// Kernel
	float *d_kernel;
	float *d_kernel_phi;
	// Images
	float *d_imgIn;
	float *d_imgOut;
	// Gradients
	float *d_gradx;
	float *d_grady;
	// Gradients for structure tensor
	float *d_gradx_tensor;	
	float *d_grady_tensor;
	float *d_m1;
	float *d_m2;
	float *d_m3;
	float *d_t1;
	float *d_t2;
	float *d_t3;
	// Norm
	float *d_norm;
	// Divergence
	float *d_div;

	////////////////////////////////////////////////////////////////////// Arrays (Host) /////////////////////////////////////////////////////////////////////

	// Kernel
	float *kernel = new float[nbytes_kernel]; 
	float *kernel_phi = new float[nbytes_kernel_phi];
	// Structure tensor
	float *m1 	 = new float[w*h];
	float *m2 	 = new float[w*h];
	float *m3 	 = new float[w*h];
	float *t1 	 = new float[w*h];
	float *t2 	 = new float[w*h];
	float *t3 	 = new float[w*h];
	// Gradient
	float *gradx = new float[nbytes];
	float *grady = new float[nbytes];
	// Divergence
	float *div   = new float[nbytes];

	////////Create kernel
	get_kernel(kernel,  w_kernel, h_kernel, pi, sigma);
	get_kernel(kernel_phi, w_kernel_phi, h_kernel_phi, pi, phi);
	// Processor type
	string processor;


	////////////////////////////////////////////////////////////////////////// CUDA	////////////////////////////////////////////////////////////////////////// 
	
	// CUDA malloc
	// Kernel
    hipMalloc(&d_kernel, nbytes_kernel);			CUDA_CHECK;
	hipMalloc(&d_kernel_phi, nbytes_kernel_phi);	CUDA_CHECK;
	// Images
    hipMalloc(&d_imgIn, nbytes); 					CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytes); 					CUDA_CHECK;
	// Gradients
	hipMalloc(&d_gradx, nbytes);					CUDA_CHECK;
	hipMalloc(&d_grady, nbytes);					CUDA_CHECK;
	// Gradients for structure tensor
	hipMalloc(&d_gradx_tensor, nbytes);			CUDA_CHECK;
	hipMalloc(&d_grady_tensor, nbytes);			CUDA_CHECK;
	hipMalloc(&d_m1, w*h*sizeof(float));			CUDA_CHECK;
	hipMalloc(&d_m2, w*h*sizeof(float));			CUDA_CHECK;
	hipMalloc(&d_m3, w*h*sizeof(float));			CUDA_CHECK;
	hipMalloc(&d_t1, w*h*sizeof(float));			CUDA_CHECK;
	hipMalloc(&d_t2, w*h*sizeof(float));			CUDA_CHECK;
	hipMalloc(&d_t3, w*h*sizeof(float));			CUDA_CHECK;
	// Norm
	hipMalloc(&d_norm, w*h*sizeof(float));			CUDA_CHECK;
	// Divergence
	hipMalloc(&d_div,   nbytes);					CUDA_CHECK;

	// CUDA copy
    hipMemcpy(d_kernel, kernel, nbytes_kernel, hipMemcpyHostToDevice);												CUDA_CHECK;
	hipMemcpy(d_kernel_phi, kernel_phi, nbytes_kernel_phi, hipMemcpyHostToDevice);									CUDA_CHECK;
    hipMemcpy(d_imgIn, imgIn, nbytes, hipMemcpyHostToDevice);			    											CUDA_CHECK;

	// Iterations
	for (size_t i = 0; i < N; i++){
		// Initial convolution - structure tensor
		convolution_global <<< grid, block >>> (d_imgIn, d_imgOut, d_kernel, w, h, nc, w_kernel, h_kernel);					CUDA_CHECK;
		// Compute gradient of the convoluted image	- structure tensor
		rotational_gradient <<< grid, block >>> (d_imgIn, d_gradx_tensor, d_grady_tensor, w, h, nc);						CUDA_CHECK;
		// Compute m1, m2, and m3 - structure tensor
		compute_M <<< grid, block >>> (d_m1, d_m2, d_m3, d_gradx_tensor, d_grady_tensor, w, h, nc);							CUDA_CHECK;
		// Convolution on m1 - structure tensor
		convolution_global <<< grid, block >>> (d_m1, d_t1, d_kernel_phi, w, h, 1, w_kernel_phi, h_kernel_phi);						CUDA_CHECK;
		// Convolution on m2 - structure tensor
		convolution_global <<< grid, block >>> (d_m2, d_t2, d_kernel_phi, w, h, 1, w_kernel_phi, h_kernel_phi);						CUDA_CHECK;
		// Convolution on m3 - structure tensor
		convolution_global <<< grid, block >>> (d_m3, d_t3, d_kernel_phi, w, h, 1, w_kernel_phi, h_kernel_phi);						CUDA_CHECK;
		// Compute gradient
		compute_gradient <<< grid, block >>> (d_gradx, d_grady, d_imgIn, w, h, nc);													CUDA_CHECK;
		// Apply diffusion tensor
		apply_diffusion <<< grid, block >>> (d_gradx, d_grady, d_imgIn, alpha, C, d_t1, d_t2, d_t3, w, h, nc);				CUDA_CHECK;
		// Compute divergence
		compute_divergence <<< grid, block >>> (d_div, d_gradx, d_grady, w, h, nc);											CUDA_CHECK;
		// Update image
		update_image <<< grid, block >>> (d_imgIn, d_div, tau, w, h, nc);													CUDA_CHECK;
	}

	// Copy the results to host
	hipMemcpy(imgOut, d_imgIn, nbytes, hipMemcpyDeviceToHost);		CUDA_CHECK;
	hipMemcpy(gradx,  d_gradx_tensor, nbytes, hipMemcpyDeviceToHost);		CUDA_CHECK;
	hipMemcpy(grady,  d_grady_tensor, nbytes, hipMemcpyDeviceToHost);		CUDA_CHECK;
	hipMemcpy(div,    d_div,   nbytes, hipMemcpyDeviceToHost);		CUDA_CHECK;

	hipMemcpy(t1, d_t1, w*h*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(t2, d_t2, w*h*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(t3, d_t3, w*h*sizeof(float), hipMemcpyDeviceToHost);



 	// Free memory
    hipFree(d_imgIn);  		CUDA_CHECK;
    hipFree(d_imgOut); 		CUDA_CHECK;
    hipFree(d_kernel); 		CUDA_CHECK;
	hipFree(d_div);			CUDA_CHECK;
	hipFree(d_gradx);  		CUDA_CHECK;
	hipFree(d_grady);  		CUDA_CHECK;
	hipFree(d_norm);			CUDA_CHECK;

	hipFree(d_t1);
	hipFree(d_t2);
	hipFree(d_t3);


	// Type of processor
	processor = "GPU - global memory";
	cout << processor << endl;



	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	timer.end();  float t = timer.get();
	cout << "time: " << t*1000 << " ms" << endl;


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)
    // show output image: first convert to interleaved opencv format from the layered raw array

	convert_layered_to_mat(mOut, imgOut);					
	showImage("Diffusion", mOut, 100+w+40, 100);


    // ### Display your own output images here as needed

	convert_layered_to_mat(mgradx, gradx);
	convert_layered_to_mat(mgrady, grady);
	convert_layered_to_mat(mdiv, div);

	convert_layered_to_mat(mt1, t1);
	convert_layered_to_mat(mt2, t2);
	convert_layered_to_mat(mt3, t3);

	showImage("t1", 10.f*mt1, 50, 250);
	showImage("t2", 10.f*mt2, 50 + w, 250);
	showImage("t3", 10.f*mt3, 50 + 2 * w, 250);

	//showImage("grad_x", mgradx, 100+w+50, 150);
	//showImage("grad_y", mgrady, 100+w+60, 150);
	//showImage("div", mdiv, 100+w+80, 200);


/*
	showImage("m1", 10.f*mM1, 50, 200);
	showImage("m2", 10.f*mM2, 50 + w, 200);
	showImage("m3", 10.f*mM3, 50 + 2 * w, 200);
	showImage("t1", 10.f*mT1, 50, 250);
	showImage("t2", 10.f*mT2, 50 + w, 250);
	showImage("t3", 10.f*mT3, 50 + 2 * w, 250);
	*/


#ifdef CAMERA
    // end of camera loop
	}
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

	// free allocated arrays
#ifdef CAMERA
	delete[] imgIn;
	delete[] imgOut;
#else
	delete[] imgIn;
	delete[] imgOut;
	delete[] kernel;
	delete[] gradx;
	delete[] grady;

#endif

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
