#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

// Exercise 4

// Written by: Jiho Yang (M.Sc student in Computational Science & Engineering)
// Matriculation number: 03675799

#include "helper.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA

// Compute gradient
__global__ void compute_gradient(float *d_gradx, float *d_grady, float *d_imgIn, int w, int h, int sizeImg){
	// Get x y z pixel coordinates in 3D kernel
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int z = threadIdx.z + blockIdx.z*blockDim.z;
	// Get high indices
	size_t x_high = x + 1 + (size_t)w*y + (size_t)h*w*z;
	size_t y_high = x + (size_t)w*(y+1) + (size_t)h*w*z;
	size_t idx = x + (size_t)w*y + (size_t)h*w*z;
	// Ensure no threads are out of problem domain
	if (idx < sizeImg){
	// Compute gradient
		if (x < w-1){
			d_gradx[idx] = d_imgIn[x_high] - d_imgIn[idx];
		} else
			d_gradx[idx] = 0;
		if (y < h-1){
			d_grady[idx] = d_imgIn[y_high] - d_imgIn[idx];
		} else
			d_grady[idx] = 0;
	}
}

// Compute divergence
__global__ void compute_divergence(float *d_div, float *d_v_1, float *d_v_2, int w, int h, int sizeImg){
	// Get x y z pixel coordinates in 3D kernel
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int z = threadIdx.z + blockIdx.z*blockDim.z;
	// Get low indices
	size_t idx = x + (size_t)w*y + (size_t)h*w*z;
	size_t x_low = x-1 + (size_t)w*y + (size_t)h*w*z;
	size_t y_low = x + (size_t)w*(y-1) + (size_t)h*w*z;
	// Temporary values
	float v_x, v_y;
	// Ensure no threads are out of problem domain
	if (idx < sizeImg){
		// Compute divergence
		if (x > 1){
			v_x = d_v_1[idx] - d_v_1[x_low];
		} else
			//d_v_1[idx] = 0;
			v_x = 0;
		if (y > 1){
			v_y = d_v_2[idx] - d_v_2[y_low];
		} else
			v_y = 0;
		// Sum gradients
		d_div[idx] = v_x + v_y;
	}
}

// Compute L2 norm
__global__ void compute_norm(float *d_norm, float *d_div, int w, int h, int nc, int sizeImg){
	// Temporary variable for norm
	float sqrd = 0;
	float val = 0;
	// Get coordinates
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	size_t idx = x + (size_t)w*y;
	// Compute L2 norm
	for (size_t i = 0; i < nc; i++){
		size_t idx_3d = idx + (size_t)i*w*h;
		// Ensure no threads are out of problem domain
		if (idx_3d < sizeImg){
			val = d_div[idx_3d];
			sqrd += val*val;
		}
	}
	d_norm[idx] = sqrtf(sqrd);
}



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    //cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);


    Timer timer; timer.start();
    // ###
    // ###
    // ### TODO: Main computation
    // ###
    // ###

	// Setup
	int sizeImg = (int)w*h*nc;
	int sizeNorm = (int)w*h;
	size_t nbytes = (size_t)(sizeImg)*sizeof(float);
	size_t nbytes_norm = (size_t)(sizeNorm)*sizeof(float);

	////////////////////////////////////////////// Gradient //////////////////////////////////////////////

	float *d_imgIn = NULL;
	float *d_imgOut = NULL;
	float *d_gradx = NULL;
	float *d_grady = NULL;
	hipMalloc(&d_gradx, nbytes); CUDA_CHECK;
	hipMalloc(&d_grady, nbytes); CUDA_CHECK;
	hipMalloc(&d_imgIn, nbytes); CUDA_CHECK;
	hipMalloc(&d_imgOut, nbytes); CUDA_CHECK;
	hipMemcpy(d_imgIn, imgIn, nbytes, hipMemcpyHostToDevice);	CUDA_CHECK;
	// Launch kernel
	dim3 block = dim3(128, 1, 1);
	dim3 grid = dim3((w+block.x-1)/block.x, (h+block.y-1)/block.y, (nc+block.z-1)/block.z);
	// Compute gradient
	compute_gradient <<<grid, block>>> (d_gradx, d_grady, d_imgIn, w, h, sizeImg);

	//////////////////////////////////////////// Divergence /////////////////////////////////////////////

	float *d_div = NULL;
	hipMalloc(&d_div, nbytes); CUDA_CHECK;
	// Compute divergence
	compute_divergence <<<grid, block>>> (d_div, d_gradx, d_grady, w, h, sizeImg);
	// Copy back to CPU
	//hipMemcpy(imgOut, d_div, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;

	///////////////////////////////////////////// L2 Norm //////////////////////////////////////////////

	float *d_norm = NULL;
	hipMalloc(&d_norm, nbytes_norm); CUDA_CHECK;
	// Compute L2 norm
	compute_norm <<<grid, block>>> (d_norm, d_div, w, h, nc, sizeImg);
	// Copy back to CPU
	hipMemcpy(imgOut, d_norm, nbytes_norm, hipMemcpyDeviceToHost); CUDA_CHECK;

	/////////////////////////////////////////////////////

	// Free memory
	hipFree(d_imgIn); CUDA_CHECK;
	hipFree(d_imgOut); CUDA_CHECK;
	hipFree(d_gradx); CUDA_CHECK;
	hipFree(d_grady); CUDA_CHECK;
	hipFree(d_div);	CUDA_CHECK;
	hipFree(d_norm);	CUDA_CHECK;

	/////////////////////////////////////////////////////

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;

	// show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



